#include "hip/hip_runtime.h"
#include "Collision_CUDA.cuh"
#include <iostream>


__global__ void collision_kernel(int* A, int* B, int* C, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	C[i] = A[i] + B[i];
}

void collision_kernel_call(int* A, int* B, int* C, int N, dim3 block_no, dim3 block_size)
{
	collision_kernel <<<block_no, block_size >>> (A, B, C, N);
	// check for error
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
}