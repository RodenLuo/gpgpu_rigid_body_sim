#include "hip/hip_runtime.h"
#include "Collision_CUDA.cuh"
#include <iostream>


__global__ void collision_kernel(float* positions, float* velocities, int numberOfBalls,
	float boxSize, float resistance, glm::vec3 gravity, int ballCollisionRun,
	glm::vec3 barrierShift, int barrierIsOn)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	/*float3 tmp = float3(0.0f, 0.0f, 0.0f);*/

	positions[3 * i] = 0;

}


void collision_kernel_call(float* positions, float* velocities, int numberOfBalls,
	float boxSize, float resistance, glm::vec3 gravity, int ballCollisionRun,
	glm::vec3 barrierShift, int barrierIsOn, dim3 grid_dim, dim3 block_dim)
{
	collision_kernel <<<grid_dim, block_dim>>> (positions, velocities, numberOfBalls,
		boxSize, resistance, gravity, ballCollisionRun,
		barrierShift, barrierIsOn);
	// check for error
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
}